#include <hip/hip_runtime.h>

/**
 * @brief returns the indices of the maximum value of all elements in the input tensor
 * 
 * @tparam T tensor data type
 * @param data input tensor
 * @param output output index tensor
 * @param n total elements number of output index tensor
 * @param stride stride of reduce dimension
 * @param dim_len reduce dimension length
 */
template<typename T>
__global__ void argmax_kernel(T* data, int* output, int n, int stride, int dim_len) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < n) {
        T max_val = data[x * stride];
        int idx = 0;
        for(int i = 1; i< dim_len; ++i) {
            if (data[(x + i) * stride] > max_val) {
                max_val = data[(x + i) * stride];
                idx = i;
            }
        }
        output[x] = idx;
    }
}

/**
 * @brief returns the indices of the maximum value of all elements in the input tensor
 * 
 * @tparam T tensor data type
 * @param data 
 * @param output 
 * @param shapes 
 * @param ndim 
 * @param dim 
 */
template<typename T>
void argmax(T* data, T* output, int* shapes, int ndim, int dim) {
    int ele_num = 1; // total output tensor element number
    for(int i = 0; i < ndim; ++i) {
        ele_num *= shapes[i];
    }
    ele_num /= shapes[dim];
    
    int stride = 1;
    for(int i = dim; i < ndim; ++i) {
        stride *= shapes[i];
    }

    int BLOCK_SIZE = 1024;
    argmax_kernel<T><<<ceil(ele_num / BLOCK_SIZE), BLOCK_SIZE>>>(data, output, ele_num, stride, shapes[dim]);
}